#include "hip/hip_runtime.h"
#include "kernels_cuda.cuh"

__host__ void cudaSafe(hipError_t error, const string msg) {
	if (error != hipSuccess) {
		cerr << "Error: " << msg << " : " << error << endl;
		exit(-1);
	}
}

__host__ void cudaCheckError(const string msg) {
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		cerr << "Error: " << msg << " : " << hipGetErrorString(error) << endl;
		exit(-1);
	}
}

// TODO: convert to cudaa
__host__  double cpu_compute_mesh_parameter(CFVMesh2D &mesh) {
	double h;
	double S;

	h = 1.e20;
	for(unsigned int cell = 0; cell < mesh.num_cells; ++cell) {
		S = mesh.cell_areas[cell];

		for(unsigned int edge = 0; edge < mesh.cell_edges_count[cell]; ++edge) {
			double length = mesh.edge_lengths[edge];
			if (h * length > S)
				h = S / length;
		}
	}

	return h;
}

__host__
void cpu_compute_edge_velocities(CFVMesh2D &mesh, CFVPoints2D<double> &velocities, CFVArray<double> &vs, double &v_max) {
	for(unsigned int i = 0; i < mesh.num_edges; ++i) {
		unsigned int left	= mesh.edge_left_cells[i];
		unsigned int right	= mesh.edge_right_cells[i];

		if (right == NO_RIGHT_CELL)
			right = left;

		double v	= ((velocities.x[left] + velocities.x[right]) * 0.5 * mesh.edge_normals.x[i])
					+ ((velocities.y[left] + velocities.y[right]) * 0.5 * mesh.edge_normals.y[i]);

		vs[i] = v;

		if (abs(v) > v_max || i == 0) {
			v_max = abs(v);
		}
	}
}

__global__
void kernel_compute_flux(CFVMesh2D_cuda *mesh, double *polution, double *velocity, double *flux, double dc) {
	// thread id = edge index
	unsigned int edge = blockIdx.x * blockDim.x + threadIdx.x;

	// check boundaries
	if (edge >= mesh->num_edges) return;

	// velocity of current edge
	double v = velocity[edge];

	if (v < 0)
		flux[edge] = v * polution[ mesh->edge_left_cells[edge] ];
	else
		flux[edge] = v * ((mesh->edge_right_cells[edge] == NO_RIGHT_CELL) ? dc : polution[ mesh->edge_right_cells[edge] ]);
}

__global__
void kernel_update(CFVMesh2D_cuda *mesh, double *polution, double *flux, double dt) {

	// thread id (cell index)
	unsigned int cell = blockIdx.x * blockDim.x + threadIdx.x;

	// check boundaries
	if (cell >= num_cells) return;

	// define start and end of neighbor edges
	unsigned int edge_limit = mesh->cell_edges_count[cell];

	// get current polution value for this cell
	double new_polution	= 0;

	// for each edge of this cell
	for(unsigned int edge_i = 0; edge_i < edge_limit; ++i) {
		unsigned int edge = mesh->cell_edges[edge_i][cell];
		// if this cell is at the left of the edge

		// amount of polution transfered through the edge
		double aux = dt * flux[edge] *
			mesh->edge_lengths[edge] /
			mesh->cell_areas[cell];

		// if this cell is on the left or the right of the edge
		if (mesh->edge_left_cells[edge] == cell) {
			new_polution -= aux;
		} else {
			new_polution += aux;
		}
	}

	// update global value
	polution[cell] += new_polution;
}


template<class T>
struct SharedMemory {
	__device__ inline operator T*() {
		extern __shared__ int __smem[];
		return (T*)__smem;
	}
	__device__ inline operator const T*() const {
		extern __shared__ int __smem[];
		return (T*)__smem;
	}
};

// specialize for double to avoid unaligned memory
// access compile errors
template<>
struct SharedMemory<double> {
	__device__ inline operator double*() {
		extern __shared__ double __smem_d[];
		return (double*)__smem_d;
	}
	__device__ inline operator const double*() const {
		extern __shared__ double __smem_d[];
		return (double*)__smem_d;
	}
};

/**
 * TODO
 * reduction - still most naive implementation
 */
template<class T, unsigned int blockSize, bool nIsPow2>
__global__
void kernel_velocities_reduction(T *g_idata, T *g_odata, unsigned int n) {

	T *sdata = SharedMemory<T>();

	// perform first level of reduction
	// reading from global memory, writing to shared memory
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockSize*2 + threadIdx.x;
	unsigned int gridSize = blockSize*2 * gridDim.x;

	T myMax = g_idata[i];

	// we reduce multiple elements per thread. The number is determined by the
	// number of active thread blocks (via gridDim). More blocks will result
	// in a larger gridSize and therefore fewer elements per thread
	while(i < n) {
		if (g_idata[i] > myMax) myMax = g_idata[i];
		// ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
		if (nIsPow2 || i + blockSize < n)
			if (g_idata[i+blockSize]) myMax = g_idata[i+blockSize];
		i += gridSize;
	}

	// each thread puts its local sum into shared memory
	sdata[tid] = myMax;
	__syncthreads();

	// do reduction in shared mem
	if (blockSize >= 512) { if (tid < 256) { if (sdata[tid+256] > myMax) { sdata[tid] = myMax = sdata[tid+256]; } __syncthreads(); } }
	if (blockSize >= 256) { if (tid < 128) { if (sdata[tid+128] > myMax) { sdata[tid] = myMax = sdata[tid+128]; } __syncthreads(); } }
	if (blockSize >= 128) { if (tid <  64) { if (sdata[tid+ 64] > myMax) { sdata[tid] = myMax = sdata[tid+ 64]; } __syncthreads(); } }

	// now that we are using warp-synchronous programming (below)
	// we need to declare our shared memory volatile so that the compiler
	// doesn't reorder stores to it and indice incorrect behavior.
	volatile T* smem = sdata;
	if (blockSize >= 64)  { if (tid <  32)  { if (smem[tid+ 32] > myMax) { smem[tid]  = myMax = smem[tid+  32]; } __syncthreads(); } }
	if (blockSize >= 32)  { if (tid <  16)  { if (smem[tid+ 16] > myMax) { smem[tid]  = myMax = smem[tid+  16]; } __syncthreads(); } }
	if (blockSize >= 32)  { if (tid <   8)  { if (smem[tid+  8] > myMax) { smem[tid]  = myMax = smem[tid+   8]; } __syncthreads(); } }
	if (blockSize >= 32)  { if (tid <   4)  { if (smem[tid+  4] > myMax) { smem[tid]  = myMax = smem[tid+   4]; } __syncthreads(); } }
	if (blockSize >= 32)  { if (tid <   2)  { if (smem[tid+  2] > myMax) { smem[tid]  = myMax = smem[tid+   2]; } __syncthreads(); } }
	if (blockSize >= 32)  { if (tid <   1)  { if (smem[tid+  1] > myMax) { smem[tid]  = myMax = smem[tid+   1]; } __syncthreads(); } }
	//if (blockSize >= 32) { if (smem[tid+16] > myMax) { smem[tid] = myMax = smem[tid+16]; } }
	//if (blockSize >= 16) { if (smem[tid+ 8] > myMax) { smem[tid] = myMax = smem[tid+ 8]; } }
	//if (blockSize >=  8) { if (smem[tid+ 4] > myMax) { smem[tid] = myMax = smem[tid+ 4]; } }
	//if (blockSize >=  4) { if (smem[tid+ 2] > myMax) { smem[tid] = myMax = smem[tid+ 2]; } }
	//if (blockSize >=  2) { if (smem[tid+ 1] > myMax) { smem[tid] = myMax = smem[tid+ 1]; } }

	// write result for this block to global mem
	if (tid == 0)
		g_odata[blockIdx.x] = sdata[0];
}

bool ispow2(unsigned int x) {
	return ((x & (x-1)) == 0);
}

unsigned int nextPow2(unsigned int x) {
	--x;
	x |= x >> 1;
	x |= x >> 2;
	x |= x >> 4;
	x |= x >> 8;
	x |= x >> 16;
	return ++x;
}

bool isPow2(unsigned int x) {
	return ((x & (x-1)) == 0);
}

#ifndef MIN
#define MIN(x,y) ((x < y) ? x : y)
#endif

void get_reduction_num_blocks_and_threads(int n, int maxBlocks, int maxThreads, int &blocks, int &threads) {
	threads = (n < maxThreads * 2) ? nextPow2((n + 1) / 2) : maxThreads;
	blocks =  (n + (threads * 2- 1)) / (threads * 2);

	// TODO this was deleted. make sure it is safe
	//blocks = MIN(maxBlocks, blVocks);
}

template<class T>
void wrapper_reduce_velocities(int size, int threads, int blocks, T *d_idata, T *d_odata) {
	dim3 dimBlock(threads, 1, 1);
	dim3 dimGrid(blocks, 1, 1);

	// when there is only one warp per block, we need to allocate two warps
	// worth of shared memory so that we don't index shared memory out of bounds
	int smemSize = (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);
	//cout << "shared size: " << smemSize << endl;

	if (isPow2(size)) {
		switch(threads) {
			case 512: kernel_velocities_reduction<T, 512, true><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case 256: kernel_velocities_reduction<T, 256, true><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case 128: kernel_velocities_reduction<T, 128, true><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case  64: kernel_velocities_reduction<T,  64, true><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case  32: kernel_velocities_reduction<T,  32, true><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case  16: kernel_velocities_reduction<T,  16, true><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case   8: kernel_velocities_reduction<T,   8, true><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case   4: kernel_velocities_reduction<T,   4, true><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case   2: kernel_velocities_reduction<T,   2, true><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case   1: kernel_velocities_reduction<T,   1, true><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
		}
	}
	else {
		switch(threads) {
			case 512: kernel_velocities_reduction<T, 512, false><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case 256: kernel_velocities_reduction<T, 256, false><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case 128: kernel_velocities_reduction<T, 128, false><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case  64: kernel_velocities_reduction<T,  64, false><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case  32: kernel_velocities_reduction<T,  32, false><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case  16: kernel_velocities_reduction<T,  16, false><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case   8: kernel_velocities_reduction<T,   8, false><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case   4: kernel_velocities_reduction<T,   4, false><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case   2: kernel_velocities_reduction<T,   2, false><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case   1: kernel_velocities_reduction<T,   1, false><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
		}
	}
}

// Instantiate reduction function for 3 types
template void wrapper_reduce_velocities<double>(int size, int threads, int blocks, double *d_idata, double *d_odata);
template void wrapper_reduce_velocities<float> (int size, int threads, int blocks, float  *d_idata, float  *d_odata);
template void wrapper_reduce_velocities<int>   (int size, int threads, int blocks, int    *d_idata, int    *d_odata);


