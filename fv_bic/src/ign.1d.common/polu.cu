#include "hip/hip_runtime.h"
#include "FVL/FVLib.h"
using namespace std;

#ifdef NO_CUDA
#include "kernels_cpu.h"
#else
#include <hip/hip_runtime.h>
#include "kernels_cuda.cuh"
#endif

#define BLOCK_SIZE_FLUX		512
#define BLOCK_SIZE_UPDATE	512
#define BLOCK_SIZE 512
#define GRID_SIZE(elems, threads)	((int) std::ceil((double)elems/threads))


typedef struct _parameters {
	string mesh_file;
	string velocity_file;
	string initial_file;
	string output_file;
	double final_time;
	double anim_time;
	int anim_jump;
	double dirichlet;
	double CFL;
} Parameters;


#define _USE_MATH_DEFINES
#include <math.h>
#include <limits>
#include <set>

void prepare_mesh_test_data(CFVMesh2D &mesh, CFVArray<double> &polution) {
	double min_x = std::numeric_limits<double>::max();
	double max_x = std::numeric_limits<double>::min();

	/* find min and max x coords of the mesh edges */
	for(unsigned int i = 0; i < mesh.num_edges; ++i) {
		double current = mesh.edge_centroids.x[i];
		if (current < min_x) min_x = current;
		if (current > max_x) max_x = current;
	}

	cout << endl << "Linking mesh ends" << endl;
	/* This assumes the mesh is rectangular, and we want to connect the left side with the right side
	 * that is, for every edge E with x = min_x, and no right cell, make the right cell equal to the left cell of the corresponding edge on the right side, and vice-versa
	 **/
	set<unsigned int> left_cells;
	set<unsigned int> right_cells;

	for(unsigned int i = 0; i < mesh.num_edges; ++i) {
		if (mesh.edge_types[i] == FV_EDGE_DIRICHLET) {
			if (mesh.edge_centroids.x[i] == min_x)
				left_cells.insert(i);
			
			if (mesh.edge_centroids.x[i] == max_x)
				right_cells.insert(i);
		}
	}

	set<unsigned int>::iterator left_it, right_it;
	for(left_it = left_cells.begin(), right_it = right_cells.begin();
		left_it != left_cells.end();
		++left_it, ++right_it) {

		unsigned int l = *left_it;
		unsigned int r = *right_it;

		/* set edges type to regular */
		mesh.edge_types[l] = FV_EDGE_FAKE;
		mesh.edge_types[r] = FV_EDGE_FAKE;

		/* link both edges */
		cout << "linking " << l << " with " << r << endl; 
		mesh.edge_right_cells[l] = mesh.edge_left_cells[l];
		mesh.edge_left_cells[l]  = mesh.edge_left_cells[r];

		mesh.edge_right_cells[r] = mesh.edge_right_cells[l];
		cout << "linking edge " << l << " with " << r << endl;
	}

	cout << "Linked " << left_cells.size() << " pairs of edges " << endl << endl;
}

Parameters read_parameters (string parameters_filename) {
	Parameters data;
	FVParameters para(parameters_filename);

	data.mesh_file		= para.getString("MeshName");
	data.velocity_file	= para.getString("VelocityFile");
	data.initial_file	= para.getString("PoluInitFile");
	data.output_file	= para.getString("OutputFile");
	data.final_time		= para.getDouble("FinalTime");
	data.anim_time		= para.getDouble("AnimTimeStep");
	data.anim_jump		= para.getInteger("NbJump");
	data.dirichlet		= para.getDouble("DirichletCondition");
	data.CFL			= para.getDouble("CFL");

	return data;
}


int main(int argc, char **argv) {

	#ifdef NO_CUDA
		cout << "Running in NO_CUDA mode" << endl;
	#endif

	// var declaration
	int i = 0;
	double h, t, dt, v_max = 0;
	string name;

	// read params
	Parameters data;
	if (argc != 2) {
		cerr << "Arg warning: no xml param filename specified. Defaulting to param.xml" << endl;
		data = read_parameters("param.xml");
	} else
		data = read_parameters(argv[1]);

	// read mesh
	FVL::CFVMesh2D		mesh(data.mesh_file);
	FVL::CFVRecons2D	recons(mesh);

	FVL::CFVPoints2D<double> velocities(mesh.num_cells);
	FVL::CFVArray<double>    polution(mesh.num_cells);
	FVL::CFVArray<double>    vs(mesh.num_edges);
#if defined(_SECOND_ORDER)
	FVL::CFVArray<double>    vecA(mesh.num_cells);
#elif defined(_MUSCL)
	FVL::CFVArray<double>    p(mesh.num_cells);
#endif


	// read other input files
	FVL::FVXMLReader velocity_reader(data.velocity_file);
	FVL::FVXMLReader polu_ini_reader(data.initial_file);
	polu_ini_reader.getVec(polution, t, name);
	velocity_reader.getPoints2D(velocities, t, name);
	polu_ini_reader.close();
	velocity_reader.close();


	/* assign test value for polution */
	prepare_mesh_test_data(mesh, polution);

	FVL::FVXMLWriter polution_writer(data.output_file);
	polution_writer.append(polution, t, "polution");

	// compute velocity vector
	// TODO: Convert to CUDA
	cpu_compute_edge_velocities(mesh, velocities, vs, v_max);
	h = cpu_compute_mesh_parameter(mesh);
	dt	= data.CFL / v_max * h;

	#ifndef NO_CUDA
	// saves whole mesh to CUDA memory
	mesh.cuda_malloc();
	recons.cuda_malloc();
	polution.cuda_malloc();
	vs.cuda_malloc();
	vecA.cuda_malloc();


	// data copy
	hipStream_t stream;
	hipStreamCreate(&stream);

	mesh.cuda_save(stream);
	polution.cuda_save(stream);
	vs.cuda_save(stream);
	vecA.cuda_save(stream);
	
	// sizes of each kernel
	// TODO: mudar BLOCK_SIZE_FLUX para MAT_A
	dim3 block_s(BLOCK_SIZE, 1, 1);
	dim3 grid_cells(GRID_SIZE(mesh.num_cells, BLOCK_SIZE));
	dim3 grid_edges(GRID_SIZE(mesh.num_edges, BLOCK_SIZE));
	#endif


	bool finished = false;
	double anim_next_step = data.anim_time;
	cout << "dt= " << dt << endl;

	while (!finished) {
		cout << "time: " << t << "   iteration: " << i << '\r';
		
		if (t + dt > data.final_time) {
			cout << endl << "Final iteration, adjusting dt" << endl;
			dt = data.final_time - t;
			finished = true;
		}

#if defined(_SECOND_ORDER)
		// Cpu version
		#ifdef NO_CUDA
			cpu_compute_a(mesh, polution, vecA);
			cpu_compute_u(mesh, recons, polution, vecA);
			cpu_compute_flux(mesh, vs, recons);
			cpu_update(mesh, recons, polution, dt);
		#else
			// TODO
		#endif

#elif defined(_MUSCL)
		#ifdef NO_CUDA
			cpu_compute_p(mesh, polution, p);
			cpu_compute_u(mesh, recons, polution, p);
			cpu_compute_flux(mesh, vs, recons);
			cpu_update(mesh, recons, polution, dt);
		#else
			// TODO
		#endif
			
#elif defined(_MOOD)
		#ifdef NO_CUDA
			// TODO
		#else
			// TODO
		#endif
#endif

		t += dt;

		if (t >= anim_next_step) {
			#ifndef NO_CUDA
				polution.cuda_get();
			#endif

			polution_writer.append(polution, t, "polution");
			anim_next_step += data.anim_time;
		}
		++i;
	}

	polution_writer.save();
	polution_writer.close();

	#ifndef NO_CUDA
	polution.cuda_free();
	vs.cuda_free();
	vecA.cuda_free();
	recons.cuda_free();
	mesh.cuda_free();
	#endif

	cout << endl << "exiting" << endl;
}

