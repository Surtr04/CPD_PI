#include "hip/hip_runtime.h"
#include "FVL/FVLib.h"
#include "FVL/FVXMLWriter.h"
#include "FVL/FVArray.h"
#include "FVio.h"
#include "FVL/FVParameters.h"
using namespace std;

#ifdef NO_CUDA
#include "kernels_cpu.h"
#else
#include <hip/hip_runtime.h>
#include "kernels_cuda.cuh"
#endif

#define BLOCK_SIZE_FLUX		512
#define BLOCK_SIZE_UPDATE	512
#define GRID_SIZE(elems, threads)	((int) std::ceil((double)elems/threads))

typedef struct _parameters {
	string mesh_file;
	string velocity_file;
	string initial_file;
	string output_file;
	double final_time;
	double anim_time;
	int anim_jump;
	double dirichlet;
	double CFL;
} Parameters;


#define _USE_MATH_DEFINES
#include <math.h>
#include <limits>
#include <set>

void prepare_mesh_test_data(CFVMesh2D &mesh, CFVArray<double> &polution) {
	double min_x = std::numeric_limits<double>::max();
	double max_x = std::numeric_limits<double>::min();

	/* find min and max x coords of the mesh edges */
	for(unsigned int i = 0; i < mesh.num_edges; ++i) {
		double current = mesh.edge_centroids.x[i];
		if (current < min_x) min_x = current;
		if (current > max_x) max_x = current;
	}

	cout << endl << "Linking mesh ends" << endl;
	/* This assumes the mesh is rectangular, and we want to connect the left side with the right side
	 * that is, for every edge E with x = min_x, and no right cell, make the right cell equal to the left cell of the corresponding edge on the right side, and vice-versa
	 **/
	set<unsigned int> left_cells;
	set<unsigned int> right_cells;

	for(unsigned int i = 0; i < mesh.num_edges; ++i) {
		if (mesh.edge_types[i] == FV_EDGE_DIRICHLET) {
			if (mesh.edge_centroids.x[i] == min_x)
				left_cells.insert(i);
			
			if (mesh.edge_centroids.x[i] == max_x)
				right_cells.insert(i);
		}
	}

	set<unsigned int>::iterator left_it, right_it;
	for(left_it = left_cells.begin(), right_it = right_cells.begin();
		left_it != left_cells.end();
		++left_it, ++right_it) {

		unsigned int l = *left_it;
		unsigned int r = *right_it;

		/* set edges type to regular */
		mesh.edge_types[l] = FV_EDGE_FAKE;
		mesh.edge_types[r] = FV_EDGE_FAKE;

		/* link both edges */
		mesh.edge_right_cells[l] = mesh.edge_left_cells[l];
		mesh.edge_left_cells[l]  = mesh.edge_left_cells[r];

		mesh.edge_right_cells[r] = mesh.edge_right_cells[l];
		cout << "linking edge " << l << " with " << r << endl;
	}

	cout << "Linked " << left_cells.size() << " pairs of edges " << endl << endl;
}

Parameters read_parameters (string parameters_filename) {
	Parameters data;
	FVParameters para(parameters_filename);

	data.mesh_file		= para.getString("MeshName");
	data.velocity_file	= para.getString("VelocityFile");
	data.initial_file	= para.getString("PoluInitFile");
	data.output_file	= para.getString("OutputFile");
	data.final_time		= para.getDouble("FinalTime");
	data.anim_time		= para.getDouble("AnimTimeStep");
	data.anim_jump		= para.getInteger("NbJump");
	data.dirichlet		= para.getDouble("DirichletCondition");
	data.CFL			= para.getDouble("CFL");

	return data;
}

// TODO: convert to cuda
double cpu_compute_mesh_parameter(CFVMesh2D &mesh) {
	double h;
	double S;

	h = 1.e20;
	for(unsigned int cell = 0; cell < mesh.num_cells; ++cell) {
		S = mesh.cell_areas[cell];

		for(unsigned int edge = 0; edge < mesh.cell_edges_count[cell]; ++edge) {
			double length = mesh.edge_lengths[edge];
			if (h * length > S)
				h = S / length;
		}
	}

	return h;
}

void cpu_compute_edge_velocities(CFVMesh2D &mesh, CFVPoints2D<double> &velocities, CFVArray<double> &vs, double &v_max) {
	for(unsigned int i = 0; i < mesh.num_edges; ++i) {
		unsigned int left	= mesh.edge_left_cells[i];
		unsigned int right	= mesh.edge_right_cells[i];

		if (right == NO_RIGHT_CELL)		
			right = left;

		double v	= ((velocities.x[left] + velocities.x[right]) * 0.5 * mesh.edge_normals.x[i])
					+ ((velocities.y[left] + velocities.y[right]) * 0.5 * mesh.edge_normals.y[i]);

		vs[i] = v;

		if (abs(v) > v_max || i == 0) {
			v_max = abs(v);
		}

		// TODO better fix for this
		if (mesh.edge_types[i] == FV_EDGE_FAKE)
			vs[i] = 1.0;
	}
}

void cudaSafe(hipError_t error, const string msg) {
	if (error != hipSuccess) {
		cerr << "Error: " << msg << " : " << error << endl;
		exit(-1);
	}
}

void cudaCheckError(const string msg) {
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		cerr << "Error: " << msg << " : " << hipGetErrorString(error) << endl;
		exit(-1);
	}
}

int main(int argc, char **argv) {
#ifdef NO_CUDA
	cout << "Running in NO_CUDA mode" << endl;
#endif

	// var declaration
	int i = 0;
	double h, t, dt, v_max = 0;
	string name;

	// read params
	Parameters data;
	if (argc != 2) {
		cerr << "Arg warning: no xml param filename specified. Defaulting to param.xml" << endl;
		data = read_parameters("param.xml");
	} else
		data = read_parameters(argv[1]);

	// read mesh
	FVL::CFVMesh2D mesh(data.mesh_file);

	FVL::CFVPoints2D<double> velocities(mesh.num_cells);
	FVL::CFVArray<double> polution(mesh.num_cells);
	FVL::CFVArray<double> flux(mesh.num_edges);
	FVL::CFVArray<double> vs(mesh.num_edges);
	FVL::CFVMat<double> matA(3, 3, mesh.num_cells);
	FVL::CFVMat<double> vecABC(3, 1, mesh.num_cells);
	FVL::CFVMat<double> vecResult(3, 1, mesh.num_cells);
	FVL::CFVArray<double> edgePsi(mesh.num_edges);
	FVL::CFVArray<double> cellPsi(mesh.num_cells);

	// read other input files
	FVL::FVXMLReader velocity_reader(data.velocity_file);
	FVL::FVXMLReader polu_ini_reader(data.initial_file);
	polu_ini_reader.getVec(polution, t, name);
	velocity_reader.getPoints2D(velocities, t, name);
	polu_ini_reader.close();
	velocity_reader.close();

	/* assign test value for polution */
	prepare_mesh_test_data(mesh, polution);

	FVL::FVXMLWriter polution_writer(data.output_file);
	polution_writer.append(polution, t, "polution");

	// compute velocity vector
	// TODO: Convert to CUDA
	cpu_compute_edge_velocities(mesh, velocities, vs, v_max);
	h = cpu_compute_mesh_parameter(mesh);
	// TODO trocar 1.0 por parametro CFL (com valores entre 0 e 1, 0.3 para esquema de ordem 2)
	dt	= data.CFL / v_max * h;

	#ifndef NO_CUDA
	// saves whole mesh to CUDA memory
	mesh.cuda_malloc();
	polution.cuda_malloc();
	flux.cuda_malloc();
	vs.cuda_malloc();
	matA.cuda_malloc();
	vecABC.cuda_malloc();
	vecResult.cuda_malloc();
	edgePsi.cuda_malloc();
	cellPsi.cuda_malloc();

	// data copy
	hipStream_t stream;
	hipStreamCreate(&stream);

	mesh.cuda_save(stream);
	polution.cuda_save(stream);
	vs.cuda_save(stream);
	

	// sizes of each kernel
	// TODO: mudar BLOCK_SIZE_FLUX para MAT_A
	dim3 grid_matA(GRID_SIZE(mesh.num_cells, BLOCK_SIZE_FLUX), 1, 1);
	dim3 block_matA(BLOCK_SIZE_FLUX, 1, 1);

	dim3 grid_vecResult(GRID_SIZE(mesh.num_cells, BLOCK_SIZE_FLUX), 1, 1);
	dim3 block_vecResult(BLOCK_SIZE_FLUX, 1, 1);

	dim3 grid_vecABC(GRID_SIZE(mesh.num_cells, BLOCK_SIZE_FLUX), 1, 1);
	dim3 block_vecABC(BLOCK_SIZE_FLUX, 1, 1);

	dim3 grid_vecValidABC(GRID_SIZE(mesh.num_cells, BLOCK_SIZE_FLUX), 1, 1);
	dim3 block_vecValidABC(BLOCK_SIZE_FLUX, 1, 1);

	dim3 grid_flux(GRID_SIZE(mesh.num_edges, BLOCK_SIZE_FLUX), 1, 1);
	dim3 block_flux(BLOCK_SIZE_FLUX, 1, 1);

	dim3 grid_update(GRID_SIZE(mesh.num_cells, BLOCK_SIZE_UPDATE), 1, 1);
	dim3 block_update(BLOCK_SIZE_UPDATE, 1, 1);
	#endif

	#ifdef NO_CUDA
		cpu_reverseA(mesh, matA);
	#else
		kernel_compute_reverseA<<< grid_matA, block_matA >>>(mesh.cuda_get(), matA.cuda_get());
		_D(cudaCheckError("cuda[compute_reverseA]"));
	#endif

	bool finished = false;
	double anim_next_step = data.anim_time;
	cout << "dt= " << dt << endl;
	while (!finished) {
	//while(t <= data.final_time) {
		cout << "time: " << t << "   iteration: " << i << '\r';
		
		if (t + dt > data.final_time) {
			cout << endl << "Final iteration, adjusting dt" << endl;
			dt = data.final_time - t;
			finished = true;
		}

		// Cpu version
		#ifdef NO_CUDA
			cpu_vecResult(mesh, polution, vecResult, data.dirichlet);								// compute system polution coeficients for system solve
			cpu_vecABC(mesh, matA, vecResult, vecABC);												// compute (a,b,c) vector
			cpu_compute_unbounded_flux(mesh, vs, vecABC, polution, flux, edgePsi, data.dirichlet, t,dt);	// compute flux
			cpu_cellPsi(mesh, edgePsi, cellPsi);													// compute Psi bounder for each cell
			cpu_bound_flux(mesh, vs, cellPsi, polution, flux, data.dirichlet, t); 						// bound previously calculated flux using psi values
			cpu_update(mesh, polution, flux, dt); 													// update
		#else

			kernel_compute_vecResult<<< grid_vecResult, block_vecResult >>>(mesh.cuda_get(), polution.cuda_get(), vecResult.cuda_get(), data.dirichlet);
			_DEBUG {
				stringstream ss;
				ss << "cuda[compute_vecResult] i=" << i;
				cudaCheckError(ss.str());
			}
			kernel_compute_vecABC<<< grid_vecABC, block_vecABC >>>(mesh.num_cells, matA.cuda_get(), vecResult.cuda_get(), vecABC.cuda_get());
			_DEBUG {
				stringstream ss;
				ss << "cuda[compute_vectABC] i=" << i;
				cudaCheckError(ss.str());
			}
			kernel_validate_ABC<<< grid_vecValidResult, block_vecValidResult >>>(mesh.cuda_get(), vs.cuda_get(), vecABC.cuda_get(), vecValidResult.cuda_get());
			kernel_compute_flux<<< grid_flux, block_flux >>>(mesh.cuda_get(), polution.cuda_get(), vs.cuda_get(), vecABC.cuda_get(), flux.cuda_get(), data.dirichlet);
	
			_DEBUG {
				stringstream ss;
				ss << "cuda[compute_flux] i=" << i;
				cudaCheckError(ss.str());
			}
			kernel_update<<< grid_update, block_update >>>(mesh.cuda_get(), polution.cuda_get(), flux.cuda_get(), dt);
			_DEBUG {
				stringstream ss;
				ss << "cuda[update] i=" << i;
				cudaCheckError(ss.str());
			}
		#endif

	t += dt;

	if (t >= anim_next_step) {
		#ifndef NO_CUDA
		polution.cuda_get();
		#endif

		polution_writer.append(polution, t, "polution");
		anim_next_step += data.anim_time;

		/*for(unsigned int cell = 0; cell < mesh.num_cells; ++cell) {
			//double c = 2*M_PI*cos(2*M_PI*mesh.cell_centroids.x[cell]);
			//cout << "ratio = " << (c / vecABC.elem(0,0,cell)) << " vecABC[" << cell << "] = " << vecABC.elem(0,0,cell) << ", cos(x) = " << c <<  endl;
			cout << "polu[" << cell << "] = " << polution[cell] << endl;
		}*/

	}
	//if ( i == 1000) break;
	++i;
}

	polution_writer.save();
	polution_writer.close();

	#ifndef NO_CUDA
	polution.cuda_free();
	flux.cuda_free();
	vs.cuda_free();
	matA.cuda_free();
	mesh.cuda_free();
	#endif

	cout << endl << "exiting" << endl;
}

