#include "hip/hip_runtime.h"
#include "kernels.cuh"

#include "CUDA/CFVLib.h"

__global__
void kernel_compute_flux(
		unsigned int num_edges,
		double *edge_normals_x,
		double *edge_normals_y,
		unsigned int *edge_left_cells,
		unsigned int *edge_right_cells,
		double *polution,
		double *velocity_x,
		double *velocity_y,
		double *flux,
		double *vs,
		double dc) {

	// get thread id
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid >= num_edges) return;

	unsigned int i_left		= edge_left_cells[tid];
	unsigned int i_right	= edge_right_cells[tid];

	double v_left[2], v_right[2];
	double p_left, p_right;

	v_left[0]	= velocity_x[i_left];
	v_left[1]	= velocity_y[i_left];
	p_left		= polution[i_left];

	if (i_right != NO_RIGHT_EDGE) {
		v_right[0]	= velocity_x[i_right];
		v_right[1]	= velocity_y[i_right];
		p_right	 	= polution[i_right];
	} else {
		v_right[0]	= v_left[0];
		v_right[1]	= v_left[1];
		p_right		= dc;
	}

	double v	= ((v_left[0] + v_right[0]) * 0.5 * edge_normals_x[tid])
				+ ((v_left[1] + v_right[1]) * 0.5 * edge_normals_y[tid]);

	if (v < 0)
		flux[tid] = v * p_right;
	else
		flux[tid] = v * p_left;

	vs[tid] = v;
}

template<class T>
struct SharedMemory {
	__device__ inline operator T*() {
		extern __shared__ int __smem[];
		return (T*)__smem;
	}
	__device__ inline operator const T*() const {
		extern __shared__ int __smem[];
		return (T*)__smem;
	}
};

// specialize for double to avoid unaligned memory
// access compile errors
template<>
struct SharedMemory<double> {
	__device__ inline operator double*() {
		extern __shared__ double __smem_d[];
		return (double*)__smem_d;
	}
	__device__ inline operator const double*() const {
		extern __shared__ double __smem_d[];
		return (double*)__smem_d;
	}
};

/**
 * TODO
 * reduction - still most naive implementation
 */
template<class T, unsigned int blockSize, bool nIsPow2>
__global__
void kernel_velocities_reduction(T *g_idata, T *g_odata, unsigned int n) {

	T *sdata = SharedMemory<T>();

	// perform first level of reduction
	// reading from global memory, writing to shared memory
	unsigned int tid = threadIdx.x;
	unsigned int i = blockIdx.x * blockSize*2 + threadIdx.x;
	unsigned int gridSize = blockSize*2 * gridDim.x;

	T myMax = g_idata[i];

	// we reduce multiple elements per thread. The number is determined by the
	// number of active thread blocks (via gridDim). More blocks will result
	// in a larger gridSize and therefore fewer elements per thread
	while(i < n) {
		if (g_idata[i] > myMax) myMax = g_idata[i];
		// ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
		if (nIsPow2 || i + blockSize < n)
			if (g_idata[i+blockSize]) myMax = g_idata[i+blockSize];
		i += gridSize;
	}

	// each thread puts its local sum into shared memory
	sdata[tid] = myMax;
	__syncthreads();

	// do reduction in shared mem
	if (blockSize >= 512) { if (tid < 256) { if (sdata[tid+256] > myMax) { sdata[tid] = myMax = sdata[tid+256]; } __syncthreads(); } }
	if (blockSize >= 256) { if (tid < 128) { if (sdata[tid+128] > myMax) { sdata[tid] = myMax = sdata[tid+128]; } __syncthreads(); } }
	if (blockSize >= 128) { if (tid <  64) { if (sdata[tid+ 64] > myMax) { sdata[tid] = myMax = sdata[tid+ 64]; } __syncthreads(); } }

	// now that we are using warp-synchronous programming (below)
	// we need to declare our shared memory volatile so that the compiler
	// doesn't reorder stores to it and indice incorrect behavior.
	volatile T* smem = sdata;
	if (blockSize >= 64)  { if (tid <  32)  { if (smem[tid+ 32] > myMax) { smem[tid]  = myMax = smem[tid+  32]; } __syncthreads(); } }
	if (blockSize >= 32)  { if (tid <  16)  { if (smem[tid+ 16] > myMax) { smem[tid]  = myMax = smem[tid+  16]; } __syncthreads(); } }
	if (blockSize >= 32)  { if (tid <   8)  { if (smem[tid+  8] > myMax) { smem[tid]  = myMax = smem[tid+   8]; } __syncthreads(); } }
	if (blockSize >= 32)  { if (tid <   4)  { if (smem[tid+  4] > myMax) { smem[tid]  = myMax = smem[tid+   4]; } __syncthreads(); } }
	if (blockSize >= 32)  { if (tid <   2)  { if (smem[tid+  2] > myMax) { smem[tid]  = myMax = smem[tid+   2]; } __syncthreads(); } }
	if (blockSize >= 32)  { if (tid <   1)  { if (smem[tid+  1] > myMax) { smem[tid]  = myMax = smem[tid+   1]; } __syncthreads(); } }
	//if (blockSize >= 32) { if (smem[tid+16] > myMax) { smem[tid] = myMax = smem[tid+16]; } }
	//if (blockSize >= 16) { if (smem[tid+ 8] > myMax) { smem[tid] = myMax = smem[tid+ 8]; } }
	//if (blockSize >=  8) { if (smem[tid+ 4] > myMax) { smem[tid] = myMax = smem[tid+ 4]; } }
	//if (blockSize >=  4) { if (smem[tid+ 2] > myMax) { smem[tid] = myMax = smem[tid+ 2]; } }
	//if (blockSize >=  2) { if (smem[tid+ 1] > myMax) { smem[tid] = myMax = smem[tid+ 1]; } }

	// write result for this block to global mem
	if (tid == 0)
		g_odata[blockIdx.x] = sdata[0];
}

bool ispow2(unsigned int x) {
	return ((x & (x-1)) == 0);
}

unsigned int nextPow2(unsigned int x) {
	--x;
	x |= x >> 1;
	x |= x >> 2;
	x |= x >> 4;
	x |= x >> 8;
	x |= x >> 16;
	return ++x;
}

bool isPow2(unsigned int x) {
	return ((x & (x-1)) == 0);
}

#ifndef MIN
#define MIN(x,y) ((x < y) ? x : y)
#endif

void get_reduction_num_blocks_and_threads(int n, int maxBlocks, int maxThreads, int &blocks, int &threads) {
	threads = (n < maxThreads * 2) ? nextPow2((n + 1) / 2) : maxThreads;
	blocks =  (n + (threads * 2- 1)) / (threads * 2);

	// TODO this was deleted. make sure it is safe
	//blocks = MIN(maxBlocks, blVocks);
}

template<class T>
void wrapper_reduce_velocities(int size, int threads, int blocks, T *d_idata, T *d_odata) {
	dim3 dimBlock(threads, 1, 1);
	dim3 dimGrid(blocks, 1, 1);

	// when there is only one warp per block, we need to allocate two warps
	// worth of shared memory so that we don't index shared memory out of bounds
	int smemSize = (threads <= 32) ? 2 * threads * sizeof(T) : threads * sizeof(T);
	//cout << "shared size: " << smemSize << endl;

	if (isPow2(size)) {
		switch(threads) {
			case 512: kernel_velocities_reduction<T, 512, true><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case 256: kernel_velocities_reduction<T, 256, true><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case 128: kernel_velocities_reduction<T, 128, true><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case  64: kernel_velocities_reduction<T,  64, true><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case  32: kernel_velocities_reduction<T,  32, true><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case  16: kernel_velocities_reduction<T,  16, true><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case   8: kernel_velocities_reduction<T,   8, true><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case   4: kernel_velocities_reduction<T,   4, true><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case   2: kernel_velocities_reduction<T,   2, true><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case   1: kernel_velocities_reduction<T,   1, true><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
		}
	}
	else {
		switch(threads) {
			case 512: kernel_velocities_reduction<T, 512, false><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case 256: kernel_velocities_reduction<T, 256, false><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case 128: kernel_velocities_reduction<T, 128, false><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case  64: kernel_velocities_reduction<T,  64, false><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case  32: kernel_velocities_reduction<T,  32, false><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case  16: kernel_velocities_reduction<T,  16, false><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case   8: kernel_velocities_reduction<T,   8, false><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case   4: kernel_velocities_reduction<T,   4, false><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case   2: kernel_velocities_reduction<T,   2, false><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
			case   1: kernel_velocities_reduction<T,   1, false><<< dimGrid, dimBlock, smemSize >>> (d_idata, d_odata, size); break;
		}
	}
}

// Instantiate reduction function for 3 types
template void wrapper_reduce_velocities<double>(int size, int threads, int blocks, double *d_idata, double *d_odata);
template void wrapper_reduce_velocities<float> (int size, int threads, int blocks, float  *d_idata, float  *d_odata);
template void wrapper_reduce_velocities<int>   (int size, int threads, int blocks, int    *d_idata, int    *d_odata);


/**
 * Update kernel
 */
__global__
void kernel_update(
		unsigned int num_edges,
		unsigned int *edge_left_cells,
		unsigned int *edge_right_cells,
		double *lengths,
		double *areas,
		double *polution,
		double *flux,
		double dt) {

	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= num_edges) return;

	unsigned int left	= edge_left_cells[tid];
	unsigned int right	= edge_right_cells[tid];

	polution[left] =
		dt * flux[tid] * lengths[tid] / areas[left];

	if (right == NO_RIGHT_EDGE)
		return;

	polution[right] =
		dt * flux[tid] * lengths[tid] / areas[right];
}

__global__
void kernel_update(
		unsigned int num_cells,
		unsigned int num_total_edges,
		unsigned int *edge_left_cells,
		unsigned int *edge_right_cells,
		double *edge_lengths,
		double *cell_areas,
		unsigned int *cell_edges,
		unsigned int *cell_edges_index,
		unsigned int *cell_edges_count,
		double *polution,
		double *flux,
		double dt) {

	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid > num_cells) return;

	// define start and end of neighbor edges
	unsigned int edge_index = cell_edges_index[tid];
	unsigned int edge_limit = edge_index + cell_edges_count[tid];

	// get current polution value for this cell
	unsigned int new_polution	= 0;

	// for each edge of this cell
	for(unsigned int i = edge_index; i < edge_limit; ++i) {
		unsigned int edge = cell_edges[i];
		// if this cell is at the left of the edge

		//double aux_polution = dt;//dt * flux[edge] * edge_lengths[edge] / cell_areas[tid];
		if (edge_left_cells[edge] == tid) {
			new_polution += 1;
		//}// else if (edge_right_cells[edge] == tid){ //otherwise, this cell is obviosly to the right of the edge
		//	new_polution += aux_polution;
		} else {
			new_polution += 10;
		}
	}

	// update global value
	polution[tid] += new_polution;
}
