#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cutil.h>

#include "CUDA/CFVLib.h"

__host__ void cuda_main_loop(
		double final_time,
		unsigned jump_interval,
		CudaFV::CFVMesh2D &mesh,
		double mesh_parameter,
		FVVect<double> &old_polution,
		CudaFV::CFVVect<double> &polutions,
		CudaFV::CFVPoints2D &velocities, CudaFV::CFVVect<double> &flux,
		double dc);

__global__ void cuda_compute_flux(...);
__global__ void cuda_reduce(...);
__global__ void cuda_update(...);




__host__ void cuda_main_loop(
		double final_time,
		unsigned jump_interval,
		CudaFV::CFVMesh2D &mesh,
		double mesh_parameter,
		FVVect<double> &old_polution,
		CudaFV::CFVVect<double> &polutions,
		CudaFV::CFVPoints2D &velocities, CudaFV::CFVVect<double> &flux,
		double dc) {

	double t, dt;
	int i;
	FVio polution_file
}
